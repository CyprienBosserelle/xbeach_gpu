#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////
//XBeach_GPU                                                                    //
//Copyright (C) 2013 Bosserelle                                                 //
//                                                                              //
//This program is free software: you can redistribute it and/or modify          //
//it under the terms of the GNU General Public License as published by          //
//the Free Software Foundation.                                                 //
//                                                                              //
//This program is distributed in the hope that it will be useful,               //
//but WITHOUT ANY WARRANTY; without even the implied warranty of                //    
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                 //
//GNU General Public License for more details.                                  //
//                                                                              //
//You should have received a copy of the GNU General Public License             //
//along with this program.  If not, see <http://www.gnu.org/licenses/>.         //
//////////////////////////////////////////////////////////////////////////////////

#include <stdio.h>


#define pi 3.14159265


__global__ void ubnd(int nx, int ny, DECNUM dx, DECNUM dt, DECNUM g, DECNUM rho, DECNUM totaltime, DECNUM wavbndtime, DECNUM rt, DECNUM slbndtime, DECNUM rtsl, DECNUM zsbndold, DECNUM zsbndnew, DECNUM Trep, DECNUM * qbndold, DECNUM * qbndnew, DECNUM *zs, DECNUM * uu, DECNUM * vv, DECNUM *vu, DECNUM * umean, DECNUM * vmean, DECNUM * zb, DECNUM * cg, DECNUM * hum, DECNUM * zo, DECNUM *Fx, DECNUM *hh)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int xplus2 = pplus2(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		DECNUM ui, vi, thetai, vert;
		DECNUM beta, betar, betat, betab, bnp1, bn;
		DECNUM ht, htr;
		DECNUM theta0 = 0.0f;
		DECNUM alpha2 = -1.0f*theta0;
		DECNUM epsi = 0.005; //Not used!
		DECNUM ur, uumean, vvmean, urr, alphanew;
		DECNUM dbetadx, dbetady, dvudy, dhdx;
		DECNUM qx, qy, zsbnd;
		DECNUM order = 2.0f;
		DECNUM ccg = cg[i];
		DECNUM cats = 4; // number of wave period to average the current from
		DECNUM factime = 0.0f;// 1.0f/cats/Trep*dt;
		DECNUM taper = min(totaltime / 100.0f, 1.0f);

		if (ix == 0)
		{

			qx = (qbndold[iy] + (totaltime - wavbndtime + rt)*(qbndnew[iy] - qbndold[iy]) / rt)*taper;
			qy = (qbndold[iy + ny] + (totaltime - wavbndtime + rt)*(qbndnew[iy + ny] - qbndold[iy + ny]) / rt)*taper;
			zsbnd = zsbndold + (totaltime - rtsl)*(zsbndnew - zsbndold) / (slbndtime - rtsl);

			ht = zsbnd + zb[i];
			htr = zsbnd + zb[xplus + iy*nx];
			ui = qx / ht;
			vi = qy / ht;
			beta = uu[i] - 2.0f*sqrt(g*hum[i]);
			betar = uu[xplus + iy*nx] - 2.0f*sqrtf(g*hum[xplus + iy*nx]);
			betat = uu[ix + yplus*nx] - 2.0f*sqrtf(g*hum[ix + yplus*nx]);
			betab = uu[ix + yminus*nx] - 2.0f*sqrtf(g*hum[ix + yminus*nx]);

			dvudy = (vu[ix + (yminus)*nx] - vu[ix + (yplus)*nx]) / (2.0f*dx);
			dbetadx = (betar - beta) / dx;
			dbetady = (betat - betab) / (2.0f*dx);

			dhdx = (htr - ht) / dx;

			bn = -1.0f*(uu[i] - sqrt(g*hum[i]))*dbetadx - vu[i] * dbetady + sqrtf(g*hum[i])*dvudy + 1 / rho*Fx[i] / hum[i] - zo[i] * sqrtf(uu[i] * uu[i] + vu[i] * vu[i])*uu[i] / hum[i] + g*dhdx;
			bnp1 = beta + bn*dt;

			//WARNING this should be very inefficient. Need to find a better way. possibly inside another kernel
			// not neededd when epsi ==0.0...or factime==0.0
			DECNUM uumm = 0.0f;
			DECNUM vvmm = 0.0f;
			/*for (int jj=0; jj<ny; jj++)
			{
			uumm=uumm+uu[ix+jj*nx];
			vvmm=vvmm+vv[ix+jj*nx];
			}*/



			uumean = factime*uumm + umean[iy] * (1 - factime);
			vvmean = factime*vvmm + vmean[iy] * (1 - factime);
			umean[iy] = uumean;
			vmean[iy] = vvmean;




			thetai = atanf(vi / (ui + 0.0000001f));

			vert = vu[i] - vvmean - vi;

			urr = (bnp1 - uumean + 2.0f*sqrtf(g*0.5f*(ht + htr)) - ui*(ccg*(cosf(thetai)) - sqrtf(g*0.5f*(ht + htr))) / (ccg*cosf(thetai)));

			for (int jj = 0; jj < 50; jj++)
			{
				ur = cosf(alpha2) / (cosf(alpha2) + 1.0f)*urr;
				/*if(ur==0.0f)
				{
				ur=0.0000001f;
				}*/
				alphanew = atanf(vert / (ur + 0.0000001f));
				if (alphanew > pi*0.5f)
				{
					alphanew = alphanew - pi;
				}
				if (alphanew <= -0.5f*pi)
				{
					alphanew = alphanew + pi;
				}
				if (abs(alphanew - alpha2) < 0.001f)
				{
					break;
				}
				alpha2 = alphanew;
			}


			//
			uu[i] = (order - 1.0f)*ui + ur + uumean;//2.0f*ui-(sqrtf(g/(zs[i]+zb[i]))*(zs[i]-zsbnd));;//
			zs[i] = 1.5f*((bnp1 - uu[i])*(bnp1 - uu[i]) / (4.0f*g) - 0.5f*(zb[i] + zb[xplus + iy*nx])) - 0.5f*((betar - uu[xplus + iy*nx])*(betar - uu[xplus + iy*nx]) / (4.0f*g) - 0.5f*(zb[xplus + iy*nx] + zb[xplus2 + iy*nx]));
			////
			//zsbnd+qx/(dx*dx)*dt;//

			hh[i] = zs[i] + zb[i];
			vv[i] = vv[xplus + iy*nx];
		}




		__syncthreads;
	}


}
__global__ void wlevslopes(int nx, int ny, DECNUM dx, DECNUM eps, DECNUM *zs, DECNUM * dzsdx, DECNUM *dzsdy, DECNUM*hh)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		__shared__ DECNUM zsi[16][16];
		__shared__ DECNUM zsr[16][16];
		__shared__ DECNUM zst[16][16];
		//int whi;
		//int whr;
		//int wht;


		// Should dzsdx be ==0 near dry cells?

		//whi=0;
		//whr=0;
		//wht=0;


		/*	if (hh[i]>eps)
			{
			whi=1;
			}
			if(hh[xplus+iy*nx]>eps)
			{
			whr=1;
			}
			if(hh[ix+yplus*nx]>eps)
			{
			wht=1;

			}
			*/
		zsi[tx][ty] = zs[i];
		zsr[tx][ty] = zs[xplus + iy*nx];
		zst[tx][ty] = zs[ix + yplus*nx];


		//dzsdx[i]=(zs[ix+1+iy*nx]-zs[ix-1+iy*nx])/(2*dx);
		dzsdx[i] = (zsr[tx][ty] - zsi[tx][ty]) / dx;//*whi*whr;
		dzsdy[i] = (zst[tx][ty] - zsi[tx][ty]) / dx;//*whi*wht;
	}


}
__global__ void calcuvvu(int nx, int ny, DECNUM dx, DECNUM *uu, DECNUM *vv, DECNUM *vu, DECNUM *uv, DECNUM * ust, DECNUM *thetamean, DECNUM *ueu_g, DECNUM *vev_g, DECNUM *vmageu, DECNUM *vmagev, int* wetu, int* wetv)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	int tx = threadIdx.x;
	int ty = threadIdx.y;


	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		DECNUM vsu, usu, ueu, veu, usv, vsv, vev, uev;

		__shared__ DECNUM usti[16][16];
		__shared__ DECNUM ustr[16][16];
		__shared__ DECNUM ustt[16][16];

		__shared__ DECNUM tmeani[16][16];
		__shared__ DECNUM tmeanr[16][16];
		__shared__ DECNUM tmeant[16][16];
		__shared__ int wetui[16][16];
		__shared__ int wetvi[16][16];

		usti[tx][ty] = ust[i];
		tmeani[tx][ty] = thetamean[i];
		ustr[tx][ty] = ust[xplus + iy*nx];
		tmeanr[tx][ty] = thetamean[xplus + iy*nx];
		ustt[tx][ty] = ust[ix + yplus*nx];
		tmeant[tx][ty] = thetamean[ix + yplus*nx];
		wetui[tx][ty] = wetu[i];
		wetvi[tx][ty] = wetv[i];


		// V-velocities at u-points

		vu[i] = 0.25f*(vv[ix + yminus*nx] + vv[ix + iy*nx] + vv[xplus + yminus*nx] + vv[xplus + iy*nx])*wetui[tx][ty];

		// U-velocities at v-points
		uv[i] = 0.25f*(uu[xminus + iy*nx] + uu[ix + iy*nx] + uu[xminus + yplus*nx] + uu[ix + yplus*nx])*wetvi[tx][ty];



		//Calculate V-stokes at u points
		vsu = 0.5f*(usti[tx][ty] * sinf(tmeani[tx][ty]) + ustr[tx][ty] * sinf(tmeanr[tx][ty]))*wetui[tx][ty];
		//Calculate U-stokes at u points
		usu = 0.5f*(usti[tx][ty] * cosf(tmeani[tx][ty]) + ustr[tx][ty] * cosf(tmeanr[tx][ty]))*wetui[tx][ty];
		//Calculate U-euler at u points
		ueu = uu[i] - usu;
		//Calculate V-euler at u points
		veu = vu[i] - vsu;
		vmageu[i] = sqrtf(ueu*ueu + veu*veu);
		ueu_g[i] = ueu;


		usv = 0.5f*(usti[tx][ty] * cosf(tmeani[tx][ty]) + ustt[tx][ty] * cosf(tmeant[tx][ty]))*wetvi[tx][ty];
		vsv = 0.5f*(usti[tx][ty] * sinf(tmeani[tx][ty]) + ustt[tx][ty] * sinf(tmeant[tx][ty]))*wetvi[tx][ty];
		vev = vv[i] - vsv;
		uev = uv[i] - usv;
		vmagev[i] = sqrtf(uev*uev + vev*vev);
		vev_g[i] = vev;

	}


}




__global__ void udepthmomcont(int nx, int ny, DECNUM dx, DECNUM eps, DECNUM ummn, int* wetu, DECNUM * zs, DECNUM * uu, DECNUM * hh, DECNUM *hum, DECNUM *hu, DECNUM * zb)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		DECNUM humi;
		DECNUM hui;

		__shared__ DECNUM  hhi[16][16];
		__shared__ DECNUM  hhip[16][16];
		//__shared__ DECNUM  hhjp[4][4];

		hui = hu[i];
		hhi[tx][ty] = hh[i];
		hhip[tx][ty] = hh[xplus + iy*nx];
		//hhjp[tx][ty]=hh[ix+(iy+1)*nx];
		__syncthreads;

		//Water depth in u-points do momentum equation: mean
		humi = 0.5f*(hhi[tx][ty] + hhip[tx][ty]);
		// Water depth in u-points do continuity equation: upwind





		__syncthreads;
		if (hui > eps && humi > eps)
		{
			wetu[i] = 1;
		}
		else
		{
			wetu[i] = 0;
		}


		hum[i] = max(humi, eps);
	}
}

__global__ void vdepthmomcont(int nx, int ny, DECNUM dx, DECNUM eps, DECNUM ummn, int* wetv, DECNUM * zs, DECNUM * vv, DECNUM * hh, DECNUM *hvm, DECNUM *hv, DECNUM * zb)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;


	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		DECNUM hvmi, hvi;

		__shared__ DECNUM  hhi[16][16];
		//__shared__ DECNUM  hhip[4][4];
		__shared__ DECNUM  hhjp[16][16];


		hhi[tx][ty] = hh[i];
		//hhip[tx][ty]=hh[ix+1+iy*nx];
		hhjp[tx][ty] = hh[ix + yplus*nx];
		__syncthreads;


		//Water depth in u-points do momentum equation: mean
		//hvmi=max(0.5f*(hh[i]+hh[ix+(min(iy,ny-2)+1)*nx]),eps);
		// Water depth in u-points do continuity equation: upwind

		//hvi=0.5f*(hhjp[tx][ty]-hhi[tx][ty])+hhi[tx][ty];
		hvmi = 0.5f*(hhi[tx][ty] + hhjp[tx][ty]);
		//hvm(i,j)=max(.5d0*(hh(i,j)+hh(i,min(ny,j)+1)),par%eps)  


		hvi = hv[i];


		if (hvi > eps && hvmi > eps)
		{
			wetv[i] = 1;
		}
		else
		{
			wetv[i] = 0;
		}
		hvm[i] = max(hvmi, eps);
	}
}

__global__ void depthhu(int nx, int ny, DECNUM dx, DECNUM ummn, DECNUM eps, DECNUM *hh, DECNUM * uu, DECNUM * hu, DECNUM *zs, DECNUM *zb)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		DECNUM hui = 0.0f;

		__shared__ DECNUM  hhi[16][16];
		__shared__ DECNUM  hhip[16][16];
		hhi[tx][ty] = hh[i];
		hhip[tx][ty] = hh[xplus + iy*nx];

		if (uu[i]>ummn)
		{
			//hui=hhi[tx][ty];
			hui = zs[i] - max(-1.0f*zb[i], -1.0f*zb[xplus + iy*nx]);
		}
		else
		{
			if (uu[i] < -1.0f*ummn)
			{
				//hui=hhip[tx][ty];
				hui = zs[xplus + iy*nx] - max(-1.0f*zb[i], -1.0f*zb[xplus + iy*nx]);
			}
			else
			{
				hui = max(max(zs[i], zs[xplus + iy*nx]) - max(-1.0f*zb[i], -1.0f*zb[xplus + iy*nx]), eps);
			}

		}
		//hui=0.5f*(hhip[tx][ty]+hhi[tx][ty]);
		hui = max(hui, 0.0f);
		hu[i] = hui;
	}

}

__global__ void depthhv(int nx, int ny, DECNUM dx, DECNUM ummn, DECNUM eps, DECNUM *hh, DECNUM * vv, DECNUM * hv, DECNUM *zs, DECNUM *zb)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);
		DECNUM hvi = 0.0f;

		__shared__ DECNUM  hhi[16][16];
		//__shared__ DECNUM  hhip[4][4];
		__shared__ DECNUM  hhjp[16][16];


		hhi[tx][ty] = hh[i];
		//hhip[tx][ty]=hh[ix+1+iy*nx];
		hhjp[tx][ty] = hh[ix + yplus*nx];
		__syncthreads;
		if (vv[i]>ummn)
		{
			//hvi=hhi[tx][ty];
			hvi = zs[i] - max(-1.0f*zb[i], -1.0f*zb[ix + yplus*nx]);
		}
		else
		{
			if (vv[i] < -1 * ummn)
			{
				//hvi=hhjp[tx][ty];
				hvi = zs[ix + yplus*nx] - max(-1.0f*zb[i], -1.0f*zb[ix + yplus*nx]);
			}
			else
			{
				hvi = max(max(zs[i], zs[ix + yplus*nx]) - max(-1.0f*zb[i], -1.0f*zb[ix + yplus*nx]), eps);
				//hv[i]=hvm[i];
			}
		}
		hvi = max(hvi, 0.0f);

		hv[i] = hvi;
	}
}

__global__ void ududx_adv(int nx, int ny, DECNUM dx, DECNUM * hu, DECNUM * hum, DECNUM * uu, DECNUM * ududx)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;
	DECNUM qin, uududx;

	__shared__ DECNUM uui[16][16];
	__shared__ DECNUM uur[16][16];
	__shared__ DECNUM uul[16][16];


	__shared__ DECNUM hui[16][16];




	__shared__ DECNUM humi[16][16];
	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);



		uui[tx][ty] = uu[i];
		uur[tx][ty] = uu[xplus + iy*nx];
		uul[tx][ty] = uu[xminus + iy*nx];


		hui[tx][ty] = hu[i];
		humi[tx][ty] = hum[i];



		uududx = 0.0f;
		qin = 0.5f*(hui[tx][ty] * uui[tx][ty] + hu[xminus + iy*nx] * uul[tx][ty]);
		//ududx
		if (qin > 0.0f)
		{
			uududx = uududx + qin / humi[tx][ty] * (uui[tx][ty] - uul[tx][ty]) / dx;
		}
		qin = -0.5f*(hui[tx][ty] * uui[tx][ty] + hu[xplus + iy*nx] * uur[tx][ty]);
		if (qin > 0.0f)
		{
			uududx = uududx + qin / humi[tx][ty] * (uui[tx][ty] - uur[tx][ty]) / dx;
		}
		ududx[i] = uududx;
	}
}


__global__ void ududx_adv2(int nx, int ny, DECNUM dx, DECNUM * hu, DECNUM * hum, DECNUM * uu, DECNUM * ududx)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;



	DECNUM qin, uududx;

	__shared__ DECNUM uui[16][16];
	__shared__ DECNUM uur[16][16];
	__shared__ DECNUM uur2[16][16];
	__shared__ DECNUM uul[16][16];
	__shared__ DECNUM uul2[16][16];

	__shared__ DECNUM hui[16][16];
	__shared__ DECNUM hur[16][16];
	__shared__ DECNUM hul[16][16];
	__shared__ DECNUM humi[16][16];

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xminus2 = mminus2(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int xplus2 = pplus2(ix, nx);
		uui[tx][ty] = uu[i];
		uur[tx][ty] = uu[xplus + iy*nx];
		uur2[tx][ty] = uu[xplus2 + iy*nx];
		uul[tx][ty] = uu[xminus + iy*nx];
		uul2[tx][ty] = uu[xminus2 + iy*nx];


		hui[tx][ty] = hu[i];
		hur[tx][ty] = hu[xplus + iy*nx];
		hul[tx][ty] = hu[xminus + iy*nx];
		humi[tx][ty] = hum[i];



		uududx = 0.0f;
		qin = 0.5f*(hui[tx][ty] * uui[tx][ty] + hul[tx][ty] * uul[tx][ty]);
		//ududx
		if (qin > 0.0f)
		{
			uududx = uududx + qin / humi[tx][ty] * (3 * uui[tx][ty] - 4 * uul[tx][ty] + uul2[tx][ty]) / (2 * dx);
		}
		qin = -0.5f*(hui[tx][ty] * uui[tx][ty] + hur[tx][ty] * uur[tx][ty]);
		if (qin > 0.0f)
		{
			uududx = uududx + qin / humi[tx][ty] * (3 * uui[tx][ty] - 4 * uur[tx][ty] + uur2[tx][ty]) / (2 * dx);
		}
		ududx[i] = uududx;
	}
}



__global__ void vdudy_adv(int nx, int ny, DECNUM dx, DECNUM * hv, DECNUM * hum, DECNUM * uu, DECNUM *vv, DECNUM * vdudy)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;


	DECNUM qin, vvdudy;

	__shared__ DECNUM uui[16][16];
	__shared__ DECNUM uut[16][16];
	__shared__ DECNUM uub[16][16];

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		uui[tx][ty] = uu[i];
		uut[tx][ty] = uu[ix + yplus*nx];
		uub[tx][ty] = uu[ix + yminus*nx];

		vvdudy = 0.0f;

		qin = 0.5f*(vv[ix + yminus*nx] * hv[ix + yminus*nx] + vv[xplus + yminus*nx] * hv[xplus + yminus*nx]);
		if (qin > 0.0f)
		{
			vvdudy = vvdudy + qin / hum[i] * (uui[tx][ty] - uub[tx][ty]) / dx;
		}
		qin = -0.5f*(vv[i] * hv[i] + vv[xplus + iy*nx] * hv[xplus + iy*nx]);
		if (qin > 0.0f)
		{
			vvdudy = vvdudy + qin / hum[i] * (uui[tx][ty] - uut[tx][ty]) / dx;
		}
		vdudy[i] = vvdudy;
	}

}

__global__ void vdudy_adv2(int nx, int ny, DECNUM dx, DECNUM * hv, DECNUM * hum, DECNUM * uu, DECNUM *vv, DECNUM * vdudy)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;


	DECNUM qin, vvdudy;

	__shared__ DECNUM uui[16][16];
	__shared__ DECNUM uut[16][16];
	__shared__ DECNUM uut2[16][16];
	__shared__ DECNUM uub[16][16];
	__shared__ DECNUM uub2[16][16];

	__shared__ DECNUM vvi[16][16];
	__shared__ DECNUM vvr[16][16];
	__shared__ DECNUM vvb[16][16];
	__shared__ DECNUM vvbr[16][16];
	__shared__ DECNUM hvi[16][16];
	__shared__ DECNUM hvr[16][16];
	__shared__ DECNUM hvb[16][16];
	__shared__ DECNUM hvbr[16][16];
	__shared__ DECNUM humi[16][16];

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);
		unsigned int yminus2 = mminus2(iy, ny);
		unsigned int yplus2 = pplus2(iy, ny);


		uui[tx][ty] = uu[i];
		uut[tx][ty] = uu[ix + yplus*nx];
		uub[tx][ty] = uu[ix + yminus*nx];
		uut2[tx][ty] = uu[ix + yplus2*nx];
		uub2[tx][ty] = uu[ix + yminus2*nx];

		vvi[tx][ty] = vv[i];
		vvr[tx][ty] = vv[xplus + iy*nx];
		vvb[tx][ty] = vv[ix + yminus*nx];
		vvbr[tx][ty] = vv[xplus + yminus*nx];
		hvi[tx][ty] = hv[i];
		hvr[tx][ty] = hv[xplus + iy*nx];
		hvb[tx][ty] = hv[ix + yminus*nx];
		hvbr[tx][ty] = hv[xplus + yminus*nx];
		humi[tx][ty] = hum[i];


		vvdudy = 0.0f;

		qin = 0.5f*(vvb[tx][ty] * hvb[tx][ty] + vvbr[tx][ty] * hvbr[tx][ty]);
		if (qin > 0.0f)
		{
			vvdudy = vvdudy + qin / humi[tx][ty] * (3 * uui[tx][ty] - 4 * uub[tx][ty] + uub2[tx][ty]) / (2 * dx);
		}
		qin = -0.5f*(vvi[tx][ty] * hvi[tx][ty] + vvr[tx][ty] * hvr[tx][ty]);
		if (qin > 0.0f)
		{
			vvdudy = vvdudy + qin / humi[tx][ty] * (3 * uui[tx][ty] - 4 * uut[tx][ty] + uut2[tx][ty]) / (2 * dx);
		}
		vdudy[i] = vvdudy;
	}

}



__global__ void vdvdy_adv(int nx, int ny, DECNUM dx, DECNUM * hv, DECNUM * hvm, DECNUM * vv, DECNUM * vdvdy)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	DECNUM qin, vvdvdy;
	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);



		vvdvdy = 0.0f;

		qin = 0.5f*(vv[i] * hv[i] + vv[ix + yminus*nx] * hv[ix + yminus*nx]);
		if (qin > 0.0f)
		{
			vvdvdy = vvdvdy + qin / hvm[i] * (vv[i] - vv[ix + (yminus)*nx]) / (dx);
		}
		qin = -0.5f*(hv[i] * vv[i] + hv[ix + (yplus)*nx] * vv[ix + (yplus)*nx]);
		if (qin > 0.0f)
		{
			vvdvdy = vvdvdy + qin / hvm[i] * (vv[i] - vv[ix + (yplus)*nx]) / (dx);
		}
		vdvdy[i] = vvdvdy;
	}
}

__global__ void vdvdy_adv2(int nx, int ny, DECNUM dx, DECNUM * hv, DECNUM * hvm, DECNUM * vv, DECNUM * vdvdy)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;



	__shared__ DECNUM vvi[16][16];
	__shared__ DECNUM vvb[16][16];
	__shared__ DECNUM vvb2[16][16];
	__shared__ DECNUM vvt[16][16];
	__shared__ DECNUM vvt2[16][16];
	__shared__ DECNUM hvi[16][16];
	__shared__ DECNUM hvb[16][16];
	__shared__ DECNUM hvt[16][16];
	__shared__ DECNUM hvmi[16][16];
	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);
		unsigned int yminus2 = mminus2(iy, ny);
		unsigned int yplus2 = pplus2(iy, ny);
		vvi[tx][ty] = vv[i];
		vvb[tx][ty] = vv[ix + yminus*nx];
		vvb2[tx][ty] = vv[ix + yminus2*nx];
		vvt[tx][ty] = vv[ix + yplus*nx];
		vvt2[tx][ty] = vv[ix + yplus2*nx];
		hvi[tx][ty] = hv[i];
		hvb[tx][ty] = hv[ix + yminus*nx];
		hvt[tx][ty] = hv[ix + yplus*nx];
		hvmi[tx][ty] = hvm[i];


		DECNUM qin, vvdvdy;

		vvdvdy = 0.0f;

		qin = 0.5*(vvi[tx][ty] * hvi[tx][ty] + vvb[tx][ty] * hvb[tx][ty]);
		if (qin > 0.0f)
		{
			vvdvdy = vvdvdy + qin / hvmi[tx][ty] * (3.0f*vvi[tx][ty] - 4.0f*vvb[tx][ty] + vvb2[tx][ty]) / (2 * dx);
		}
		qin = -0.5f*(hvi[tx][ty] * vvi[tx][ty] + hvt[tx][ty] * vvt[tx][ty]);
		if (qin > 0.0f)
		{
			vvdvdy = vvdvdy + qin / hvmi[tx][ty] * (3.0f*vvi[tx][ty] - 4.0f*vvt[tx][ty] + vvt2[tx][ty]) / (2 * dx);
		}
		vdvdy[i] = vvdvdy;
	}
}

__global__ void udvdx_adv(int nx, int ny, DECNUM dx, DECNUM * hu, DECNUM * hvm, DECNUM * uu, DECNUM * vv, DECNUM * udvdx)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	DECNUM qin, uudvdx;

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);



		uudvdx = 0.0f;
		qin = 0.5*(uu[xminus + iy*nx] * hu[xminus + iy*nx] + uu[xminus + yplus*nx] * hu[xminus + yplus*nx]);
		if (qin > 0.0f)
		{
			uudvdx = uudvdx + qin / hvm[i] * (vv[i] - vv[xminus + iy*nx]) / (dx);
		}
		qin = -0.5*(uu[i] * hu[i] + uu[ix + yplus*nx] * hu[ix + yplus*nx]);
		if (qin > 0.0f)
		{
			uudvdx = uudvdx + qin / hvm[i] * (vv[i] - vv[xplus + iy*nx]) / (dx);
		}

		udvdx[i] = uudvdx;
	}


}

__global__ void udvdx_adv2(int nx, int ny, DECNUM dx, DECNUM * hu, DECNUM * hvm, DECNUM * uu, DECNUM * vv, DECNUM * udvdx)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;



	__shared__ DECNUM uui[16][16];
	__shared__ DECNUM uut[16][16];
	__shared__ DECNUM uul[16][16];
	__shared__ DECNUM uutl[16][16];
	__shared__ DECNUM vvi[16][16];
	__shared__ DECNUM vvl[16][16];
	__shared__ DECNUM vvl2[16][16];
	__shared__ DECNUM vvr[16][16];
	__shared__ DECNUM vvr2[16][16];
	__shared__ DECNUM hui[16][16];
	__shared__ DECNUM hut[16][16];
	__shared__ DECNUM hul[16][16];
	__shared__ DECNUM hutl[16][16];
	__shared__ DECNUM hvmi[16][16];

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int xminus2 = mminus2(ix, nx);
		unsigned int xplus2 = pplus2(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);
		uui[tx][ty] = uu[i];
		uut[tx][ty] = uu[ix + yplus*nx];
		uul[tx][ty] = uu[xminus + iy*nx];
		uutl[tx][ty] = uu[xminus + yplus*nx];
		vvi[tx][ty] = vv[i];
		vvl[tx][ty] = vv[xminus + iy*nx];
		vvl2[tx][ty] = vv[xminus2 + iy*nx];
		vvr[tx][ty] = vv[xplus + iy*nx];
		vvr2[tx][ty] = vv[xplus2 + iy*nx];
		hui[tx][ty] = hu[i];
		hut[tx][ty] = hu[ix + yplus*nx];
		hul[tx][ty] = hu[xminus + iy*nx];
		hutl[tx][ty] = hu[xminus + yplus*nx];
		hvmi[tx][ty] = hvm[i];


		DECNUM qin, uudvdx;

		uudvdx = 0.0f;
		qin = 0.5*(uul[tx][ty] * hul[tx][ty] + uutl[tx][ty] * hutl[tx][ty]);
		if (qin > 0.0f)
		{
			uudvdx = uudvdx + qin / hvmi[tx][ty] * (3 * vvi[tx][ty] - 4 * vvl[tx][ty] + vvl2[tx][ty]) / (2 * dx);
		}
		qin = -0.5*(uui[tx][ty] * hui[tx][ty] + uut[tx][ty] * hut[tx][ty]);
		if (qin > 0.0f)
		{
			uudvdx = uudvdx + qin / hvmi[tx][ty] * (3 * vvi[tx][ty] - 4 * vvr[tx][ty] + vvr2[tx][ty]) / (2 * dx);
		}

		udvdx[i] = uudvdx;
	}

}


__global__ void smago(int nx, int ny, DECNUM dx, DECNUM * uu, DECNUM * vv, DECNUM nuh, DECNUM * nuhgrid, int usesmago)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	DECNUM dudx, dudy, dvdx, dvdy, tau;


	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		if (usesmago == 1)
		{
			dudx = (uu[i] - uu[xminus + iy*nx]) / dx;
			dudy = 0.50f*(uu[ix + yplus*nx] - uu[ix + yminus*nx] + uu[xminus + yplus*nx] - uu[xminus + yminus*nx]) / dx;
			dvdy = (vv[i] - vv[ix + yminus*nx]) / dx;
			dvdx = 0.50f*(vv[xplus + iy*nx] - vv[xminus + iy*nx] + vv[xplus + yminus*nx] - vv[xminus + yminus*nx]) / dx;
			tau = sqrt(2.0f*dudx*dudx + 2.0f*dvdy*dvdy + powf(dudy + dvdx, 2.0f));
			nuhgrid[i] = nuh*nuh*dx*dx*tau;
		}
		else
		{
			nuhgrid[i] = nuh;
		}

	}


}

__global__ void viscou(int nx, int ny, DECNUM dx, DECNUM rho, DECNUM eps, DECNUM nuhfac, DECNUM * nuhgrid, DECNUM *hh, DECNUM *hum, DECNUM *hvm, DECNUM * DR, DECNUM *uu, int * wetu, DECNUM * viscu)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;



	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		DECNUM nuh = nuhgrid[i];

		DECNUM nnuh, dudx1, dudx2, dudy1, dudy2;

		//if(ix>3)
		//{
		nnuh = max(nuh, nuhfac*hh[i] * powf(DR[i] / rho, 1.0f / 3.0f));
		//}
		//else
		//{
		//	nnuh=nuh*10;
		//}
		dudx1 = hh[xplus + iy*nx] * (uu[xplus + iy*nx] - uu[i]) / dx;
		dudx2 = hh[i] * (uu[i] - uu[xminus + iy*nx]) / dx;
		dudy1 = 0.5f*(hvm[i] + hvm[xplus + iy*nx])*(uu[ix + yplus*nx] - uu[i]) / dx;
		dudy2 = 0.5f*(hvm[ix + yminus*nx] + hvm[xplus + yminus*nx])*(uu[i] - uu[ix + yminus*nx]) / dx;
		viscu[i] = nnuh / hum[i] * ((dudx1 - dudx2) / (dx)*wetu[xplus + iy*nx] * wetu[xminus + iy*nx] + (dudy1 - dudy2) / dx*wetu[ix + yplus*nx] * wetu[ix + yminus*nx]);

		//*wetu[xplus+iy*nx]*wetu[xplus+iy*nx]
	}
}

__global__ void viscov(int nx, int ny, DECNUM dx, DECNUM rho, DECNUM eps, DECNUM nuhfac, DECNUM * nuhgrid, DECNUM *hh, DECNUM *hum, DECNUM *hvm, DECNUM * DR, DECNUM *vv, int * wetv, DECNUM * viscv)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;


	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		DECNUM nuh = nuhgrid[i];

		DECNUM nnuh, dvdx1, dvdx2, dvdy1, dvdy2;

		nnuh = max(nuh, nuhfac*hh[i] * powf(DR[i] / rho, 1.0f / 3.0f));

		dvdx1 = 0.5f*(hum[i] + hum[ix + yplus*nx])*(vv[xplus + iy*nx] - vv[i]) / dx;
		dvdx2 = 0.5f*(hum[xminus + iy*nx] + hum[xminus + yplus*nx])*(vv[i] - vv[xminus + iy*nx]) / dx;
		dvdy1 = hh[ix + yplus*nx] * (vv[ix + yplus*nx] - vv[i]) / dx;
		dvdy2 = hh[i] * (vv[i] - vv[ix + yminus*nx]) / dx;
		viscv[i] = nnuh / hvm[i] * ((dvdx1 - dvdx2) / (dx)*wetv[xplus + iy*nx] * wetv[xminus + iy*nx] + (dvdy1 - dvdy2) / dx*wetv[ix + yplus*nx] * wetv[ix + yminus*nx]);
	}
}

__global__ void viscovbnd(int nx, int ny, DECNUM * viscv)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		if (iy == ny - 1)
		{
			viscv[i] = viscv[ix + yminus*nx];
		}
		if (iy == 0)
		{
			viscv[i] = viscv[ix + yplus*nx];
		}
	}
}



__global__ void eulerustep(int nx, int ny, DECNUM dx, DECNUM dt, DECNUM g, DECNUM rho, DECNUM * zo, DECNUM fc, DECNUM windth, DECNUM windv, DECNUM Cd, DECNUM *uu, DECNUM * urms, DECNUM *ududx, DECNUM *vdudy, DECNUM *viscu, DECNUM *dzsdx, DECNUM *hu, DECNUM *hum, DECNUM *Fx, DECNUM *vu, DECNUM * ueu_g, DECNUM * vmageu, int *wetu)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	DECNUM ueu;
	DECNUM taubx;
	DECNUM hmin = 0.2;


	__shared__ DECNUM  uui[16][16];


	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);




		uui[tx][ty] = uu[i];
		ueu = ueu_g[i];

		__syncthreads;

		//&& ix>0
		if (wetu[i] == 1)
		{
			taubx = zo[i] * rho*ueu*sqrtf(1.3456f*urms[i] * urms[i] + vmageu[i] * vmageu[i]);

			uui[tx][ty] = uui[tx][ty] - dt*(ududx[i] + vdudy[i] - viscu[i] + g*dzsdx[i] + taubx / (rho*hu[i]) - Fx[i] / (rho*max(hum[i], hmin)) - 1.25f*Cd*cosf(windth)*windv*windv / (rho*hum[i]) - fc*vu[i]);

			//viscu[i]=taubx;

		}
		else
		{
			uui[tx][ty] = 0.0f;
			viscu[i] = 0.0f;

		}
		if (ix > 0)
		{
			uu[i] = uui[tx][ty];

		}
	}

}

__global__ void eulervstep(int nx, int ny, DECNUM dx, DECNUM dt, DECNUM g, DECNUM rho, DECNUM * zo, DECNUM fc, DECNUM windth, DECNUM windv, DECNUM Cd, DECNUM *vv, DECNUM * urms, DECNUM *udvdx, DECNUM *vdvdy, DECNUM *viscv, DECNUM *dzsdy, DECNUM *hv, DECNUM *hvm, DECNUM *Fy, DECNUM *uv, DECNUM * vev_g, DECNUM * vmagev, int *wetv)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	__shared__ DECNUM  vvi[16][16];
	__shared__ DECNUM  urmsi[16][16];
	__shared__ DECNUM  vmagvi[16][16];
	__shared__ DECNUM  hvmi[16][16];

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);



		DECNUM tauby, vev;

		DECNUM hmin = 0.2;

		vvi[tx][ty] = vv[i];
		urmsi[tx][ty] = urms[i];
		vmagvi[tx][ty] = vmagev[i];
		hvmi[tx][ty] = hvm[i];

		// && ix>0
		if (wetv[i] == 1)
		{
			vev = vev_g[i];

			tauby = zo[i] * rho*vev*sqrtf(1.3456f*urmsi[tx][ty] * urmsi[tx][ty] + vmagvi[tx][ty] * vmagvi[tx][ty]);
			vvi[tx][ty] = vvi[tx][ty] - dt*(udvdx[i] + vdvdy[i] - viscv[i] + g*dzsdy[i] + tauby / (rho*hv[i]) - Fy[i] / (rho*max(hvmi[tx][ty], hmin)) + fc*uv[i] - 1.25f*Cd*sinf(windth)*windv*windv / (rho*hvmi[tx][ty]));

			//viscv[i]=tauby;

		}
		else
		{
			vvi[tx][ty] = 0.0f;
			viscv[i] = 0.0f;
		}
		if (ix > 0)// && iy>0 && iy<ny)
		{
			vv[i] = vvi[tx][ty];
		}//vdvdy[i]=tauby;

	}
}




__global__ void continuity(int nx, int ny, DECNUM dx, DECNUM dt, DECNUM eps, DECNUM * uu, DECNUM* hu, DECNUM* vv, DECNUM* hv, DECNUM* zs, DECNUM *hh, DECNUM *zb, DECNUM * dzsdt)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;

	unsigned int tx = threadIdx.x;
	unsigned int ty = threadIdx.y;



	DECNUM qx, qy, qxm, qym, dzdt;
	DECNUM zz;

	__shared__ DECNUM uui[16][16];
	__shared__ DECNUM uul[16][16];
	__shared__ DECNUM vvi[16][16];
	__shared__ DECNUM vvb[16][16];
	__shared__ DECNUM hui[16][16];
	__shared__ DECNUM hul[16][16];
	__shared__ DECNUM hvi[16][16];
	__shared__ DECNUM hvb[16][16];

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		uui[tx][ty] = uu[i];
		vvi[tx][ty] = vv[i];
		uul[tx][ty] = uu[xminus + iy*nx];
		vvb[tx][ty] = vv[ix + yminus*nx];
		hui[tx][ty] = hu[i];
		hul[tx][ty] = hu[xminus + iy*nx];
		hvi[tx][ty] = hv[i];
		hvb[tx][ty] = hv[ix + yminus*nx];




		zz = zs[i];

		qx = uui[tx][ty] * hui[tx][ty];
		qy = vvi[tx][ty] * hvi[tx][ty];

		qxm = uul[tx][ty] * hul[tx][ty];

		qym = vvb[tx][ty] * hvb[tx][ty];
		dzdt = (qxm - qx + qym - qy) / dx;


		__syncthreads;

		if (ix > 0)
		{
			dzsdt[i] = dzdt;


			zs[i] = zz + dzdt*dt;

			//hh[i]=max(hh[i]+dzdt*dt,eps);
		}
	}

}

__global__ void hsbnd(int nx, int ny, DECNUM eps, DECNUM * hh, DECNUM *zb, DECNUM *zs)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	int tx = threadIdx.x;
	int ty = threadIdx.y;




	__shared__ DECNUM Fi[16][16];
	__shared__ DECNUM Ft[16][16];
	__shared__ DECNUM Fb[16][16];
	__shared__ DECNUM Fr[16][16];

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		Fi[tx][ty] = max(hh[i], eps);
		Ft[tx][ty] = max(hh[ix + yplus*nx], eps);
		Fb[tx][ty] = max(hh[ix + yminus*nx], eps);
		Fr[tx][ty] = max(hh[xplus + iy*nx], eps);

		//hh[i]=Fi[tx][ty];



		hh[i] = max(zb[i] + zs[i], eps);
	}


}

__global__ void uvlatbnd(int nx, int ny, DECNUM * vu, DECNUM * uv, DECNUM * ueu, DECNUM * vev, DECNUM * vmageu, DECNUM * vmagev)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	int tx = threadIdx.x;
	int ty = threadIdx.y;



	__shared__ DECNUM vut[16][16];
	__shared__ DECNUM vub[16][16];
	__shared__ DECNUM uvt[16][16];
	__shared__ DECNUM uvb[16][16];
	__shared__ DECNUM ueub[16][16];
	__shared__ DECNUM ueut[16][16];
	__shared__ DECNUM vevt[16][16];
	__shared__ DECNUM vevb[16][16];
	__shared__ DECNUM vmagevt[16][16];
	__shared__ DECNUM vmagevb[16][16];
	__shared__ DECNUM vmageut[16][16];
	__shared__ DECNUM vmageub[16][16];

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		uvt[tx][ty] = uv[ix + yplus*nx];
		uvb[tx][ty] = uv[ix + yminus*nx];
		vut[tx][ty] = vu[ix + yplus*nx];
		vub[tx][ty] = vu[ix + yminus*nx];
		ueut[tx][ty] = ueu[ix + yplus*nx];
		ueub[tx][ty] = ueu[ix + yminus*nx];
		vevt[tx][ty] = vev[ix + yplus*nx];
		vevb[tx][ty] = vev[ix + yminus*nx];
		vmageut[tx][ty] = vmageu[ix + yplus*nx];
		vmageub[tx][ty] = vmageu[ix + yminus*nx];
		vmagevt[tx][ty] = vmagev[ix + yplus*nx];
		vmagevb[tx][ty] = vmagev[ix + yminus*nx];

		if (iy == 0)
		{
			uv[i] = uvt[tx][ty];
			vu[i] = vut[tx][ty];
			ueu[i] = ueut[tx][ty];
			vev[i] = vevt[tx][ty];
			vmageu[i] = vmageut[tx][ty];
			vmagev[i] = vmagevt[tx][ty];
		}
		if (iy == ny - 1)
		{
			uv[i] = uvb[tx][ty];
			vu[i] = vub[tx][ty];
			ueu[i] = ueub[tx][ty];
			vev[i] = vevb[tx][ty];
			vmageu[i] = vmageub[tx][ty];
			vmagev[i] = vmagevb[tx][ty];
		}
	}
}


__global__ void uuvvzslatbnd(int nx, int ny, DECNUM * uu, DECNUM * vv, DECNUM *zs)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	int tx = threadIdx.x;
	int ty = threadIdx.y;



	__shared__ DECNUM vvr[16][16];
	__shared__ DECNUM vvb[16][16];
	__shared__ DECNUM vvt[16][16];
	__shared__ DECNUM uut[16][16];
	__shared__ DECNUM uub[16][16];
	__shared__ DECNUM zst[16][16];
	__shared__ DECNUM zsb[16][16];
	__shared__ DECNUM zsl[16][16];

	if (ix < nx && iy < ny)
	{

		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		uut[tx][ty] = uu[ix + yplus*nx];
		uub[tx][ty] = uu[ix + yminus*nx];
		vvr[tx][ty] = vv[xplus + iy*nx];
		vvt[tx][ty] = vv[ix + yplus*nx];
		vvb[tx][ty] = vv[ix + yminus*nx];
		zst[tx][ty] = zs[ix + yplus*nx];
		zsb[tx][ty] = zs[ix + yminus*nx];
		zsl[tx][ty] = zs[xminus + iy*nx];

		//F[i]=Fi[tx][ty]*wet;
		if (iy == 0)
		{
			uu[i] = uut[tx][ty];
			vv[i] = vvt[tx][ty];
			zs[i] = zst[tx][ty];
		}
		if (iy == ny - 1)
		{
			uu[i] = uub[tx][ty];
			vv[i] = vvb[tx][ty];// THis is to follow XBeach definition although I don't really agree with it
			zs[i] = zsb[tx][ty];
		}
		//		if (iy==ny-2)
		//		{
		//			vv[i]=vvb[tx][ty];// THis is to follow XBeach definition although I don't really agree with it 
		//							  // It should be that vv(i,ny-1)=vv(i,ny-2) end of story
		//		}
		if (ix == 0)
		{
			//vv[i]=vvr[tx][ty];//Imcompatible with abs_2d front boundary 
		}
		if (ix == nx - 1)
		{
			//zs[i]=zsl[tx][ty];//Need to fix 
		}
	}

}
