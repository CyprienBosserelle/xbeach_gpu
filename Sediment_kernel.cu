#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////
//XBeach_GPU                                                                    //
//Copyright (C) 2013 Bosserelle                                                 //
//                                                                              //
//This program is free software: you can redistribute it and/or modify          //
//it under the terms of the GNU General Public License as published by          //
//the Free Software Foundation.                                                 //
//                                                                              //
//This program is distributed in the hope that it will be useful,               //
//but WITHOUT ANY WARRANTY; without even the implied warranty of                //    
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                 //
//GNU General Public License for more details.                                  //
//                                                                              //
//You should have received a copy of the GNU General Public License             //
//along with this program.  If not, see <http://www.gnu.org/licenses/>.         //
//////////////////////////////////////////////////////////////////////////////////

#include <stdio.h>

#define pi 3.14159265

// declare texture reference for 2D DECNUM texture
//texture<DECNUM, 2, hipReadModeElementType> texU;
//texture<DECNUM, 2, hipReadModeElementType> texV;
//texture<DECNUM, 2, hipReadModeElementType> texZ;

__global__ void longturb(int nx, int ny,DECNUM dx, DECNUM rho,DECNUM g,DECNUM dt,DECNUM beta,DECNUM * c,DECNUM *kturb,DECNUM * rolthick,DECNUM *dzsdt,DECNUM * uu,DECNUM *vv, DECNUM *hu, DECNUM *hv,int * wetu, int * wetv,DECNUM *h)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i=ix+iy*nx;
	int tx =threadIdx.x;
	int ty= threadIdx.y;


	
	
	__shared__ DECNUM  uui[16][16];
	__shared__ DECNUM  uul[16][16];
	__shared__ DECNUM  vvi[16][16];
	__shared__ DECNUM  vvb[16][16];
	__shared__ DECNUM kturbi[16][16];
	__shared__ DECNUM kturbl[16][16];
	__shared__ DECNUM kturbr[16][16];
	__shared__ DECNUM kturbb[16][16];
	__shared__ DECNUM kturbt[16][16];

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		// use lagrangian velocities
		DECNUM kturbu = 0.0f;
		DECNUM kturbv = 0.0f;
		DECNUM dzsdt_cr = beta*c[i];
		DECNUM kkturb;
		DECNUM kturbumin, kturbvmin;
		DECNUM Sturbu, Sturbv, Sturbumin, Sturbvmin;
		DECNUM ksource, rolth;
		DECNUM betad = 1.0f;

		DECNUM hold = h[i] - dzsdt[i] * dt;

		kturbi[tx][ty] = kturb[i];
		kturbr[tx][ty] = kturb[xplus + iy*nx];
		kturbl[tx][ty] = kturb[xminus + iy*nx];
		kturbb[tx][ty] = kturb[ix + yminus*nx];
		kturbt[tx][ty] = kturb[ix + yplus*nx];

		uui[tx][ty] = uu[i];
		uul[tx][ty] = uu[xminus + iy*nx];

		vvi[tx][ty] = vv[i];
		vvb[tx][ty] = vv[ix + yminus*nx];


		// Update roller thickness
		rolth = rolthick[i] + dt*(abs(dzsdt[i]) - dzsdt_cr);
		rolthick[i] = max(rolth, 0.0f);

		//  X-direction
		kturbu = kturbi[tx][ty] * max(uui[tx][ty], 0.0f) + kturbr[tx][ty] * min(uui[tx][ty], 0.0f);
		/*if(uu[i]>0.0f)
		{
		kturbu=kturb[i];
		}
		else
		{
		if (uu[i]<0.0f)
		{     kturbu=kturb[xplus+iy*nx];}
		else
		{     kturbu=0.5f*(kturb[i]+kturb[xplus+iy*nx]);}
		}*/
		kturbumin = kturbl[tx][ty] * max(uul[tx][ty], 0.0f) + kturbi[tx][ty] * min(uul[tx][ty], 0.0f);
		/*if(uu[xminus+iy*nx]>0.0f)
		{
		kturbumin=kturb[xminus+iy*nx];
		}
		else
		{
		if(uu[xminus+iy*nx]<0.0f)
		{
		kturbumin=kturb[i];
		}
		else
		{
		kturbumin=0.5f*(kturb[xminus+iy*nx]+kturb[i]);
		}
		}*/


		Sturbu = kturbu*hu[i] * wetu[i];
		Sturbumin = kturbumin*hu[xminus + iy*nx] * wetu[xminus + iy*nx];


		// Y-direction
		kturbv = kturbi[tx][ty] * max(vvi[tx][ty], 0.0f) + kturbt[tx][ty] * min(vvi[tx][ty], 0.0f);

		/*if(vv[i]>0.0f)
		{
		kturbv=kturb[i];
		}
		else
		{
		if(vv[i]<0)
		{
		kturbv=kturb[ix+yplus*nx];
		}
		else
		{
		kturbv=0.5f*(kturb[i]+kturb[ix+yplus*nx]);
		}
		}*/
		kturbvmin = kturbb[tx][ty] * max(vvb[tx][ty], 0.0f) + kturbi[tx][ty] * min(vvb[tx][ty], 0.0f);
		/*if(vv[ix+yminus*nx]>0.0f)
		{
		kturbvmin=kturb[ix+yminus*nx];
		}
		else
		{
		if(vv[ix+yminus*nx]<0.0f)
		{
		kturbvmin=kturb[i];
		}
		else
		{
		kturbvmin=0.5f*(kturb[ix+yminus*nx]+kturb[i]);
		}
		}*/

		Sturbv = kturbv*hv[i] * wetv[i];
		Sturbvmin = kturbvmin*hv[ix + yminus*nx] * wetv[ix + yminus*nx];

		ksource = g*rolthick[i] * beta*c[i];     // only important in shallow water, where c=sqrt(gh)  

		kkturb = hold*kturb[i] - dt*((Sturbu - Sturbumin) / dx + (Sturbv - Sturbvmin) / dx - (ksource - betad*powf(kturb[i], 1.5f)));

		kturb[i] = 0.0f;//max(kkturb,0.0f);
	}


}

__global__ void Sbvr(int nx, int ny, DECNUM rho,DECNUM g,DECNUM eps, DECNUM Trep,DECNUM D50, DECNUM D90, DECNUM rhosed,DECNUM ws,DECNUM nuhfac,DECNUM * ueu, DECNUM * vev,DECNUM *H,DECNUM * DR,DECNUM * R, DECNUM * c,DECNUM * hh,DECNUM *urms,DECNUM * ceqsg,DECNUM * ceqbg, DECNUM *Tsg, DECNUM *zom, DECNUM * kturb)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i=ix+iy*nx;
	int tx =threadIdx.x;
	int ty= threadIdx.y;




	

	__shared__ DECNUM  hhi[16][16];
	//__shared__ DECNUM  Hi[16][16];
	__shared__ DECNUM  ueui[16][16];
	__shared__ DECNUM  ueul[16][16];
	__shared__ DECNUM  vevi[16][16];
	__shared__ DECNUM  vevb[16][16];
	
	DECNUM ue,ve;
	
	DECNUM vmags,vmag,ML,Tbore,dcfin,dcf,kb,Urms2;
	DECNUM B2,T1,Ucrc,Ucrw,Ucr,Ass,Asb,Cd,ceqb,ceqs;
	//DECNUM D50=0.0038;
	//DECNUM D90=0.0053;
	DECNUM zo=0.006f;//zom[i];
	DECNUM sedcal=1.0f;
	int wetz;
	DECNUM bulk=1.0f;//1.0f;


	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		//DECNUM rhosed=2500; //Sediment density
		DECNUM drho = (rhosed - rho) / rho;
		DECNUM dester = powf(drho*g, 1.0f / 3.0f) / 0.0001f*D50; //1.19e-4 comes from (Kb^2)^1/3 with Kb = 1.3e-6 m2s-2 kinematic viscosity of water
		// Calc euler velocities at cell center
		hhi[tx][ty] = max(hh[i], 0.01f);;
		ueui[tx][ty] = ueu[i];
		vevi[tx][ty] = vev[i];
		//Hi[tx][ty]=H[i];



		ueul[tx][ty] = ueu[xminus + iy*nx];
		vevb[tx][ty] = vev[ix + yminus*nx];
		__syncthreads;

		ue = 0.5*(ueul[tx][ty] + ueui[tx][ty]);
		ve = 0.5*(vevb[tx][ty] + vevi[tx][ty]);

		//need to check this...
		vmags = ue*ue + ve*ve;
		vmag = sqrt(vmags);

		//Mixing length
		ML = max(min(sqrt(2.0f*R[i] * Trep / (rho*c[i])), hhi[tx][ty]), 0.01f);

		//Bore period
		Tbore = Trep / 4.0f;// should be more complex //to improve later

		//Exponential decay of turbulence over time
		dcfin = expf(min(hhi[tx][ty], 100.0f) / ML);
		dcf = min(1 / (dcfin - 1), 1.0f);

		//Short wave turbulence (Breaking):
		kb = nuhfac*powf(DR[i] / rho, 0.66666666667f)*dcf;

		Urms2 = urms[i] * urms[i] + 1.45f*(kb + kturb[i]);//not been tested yet!!!



		DECNUM tsfac = 0.1f;
		//DECNUM ws=0.0509f;
		DECNUM Tsmin = 0.5f;
		Tsg[i] = max(tsfac*hhi[tx][ty] / ws, Tsmin); //should be different for each sediment class


		//critical U due to current
		//Ucrc=8.5f*pow(D50,0.6f)*log(4.0f*hhi[tx][ty]/D90)/log(10.0f);//Shields

		//Critical U due to Waves
		//Ucrw=0.95f*pow(1.65f*g,0.57f)*pow(D50,0.43f)*pow(Trep,0.14f);//Komar and Miller 1975

		//Critical velocity
		//Ucr=B2*Ucrc+(1.0f-B2)*Ucrw;
		if (D50 <= 0.0005f)
		{
			Ucr = 0.19f*powf(D50, 0.1f)*log10f(4.0f*hhi[tx][ty] / D90);
		}
		else
		{
			Ucr = 8.50f*powf(D50, 0.6f)*log10f(4.0f*hhi[tx][ty] / D90);

		}

		//drag coeff
		DECNUM hdrag = max(hhi[tx][ty], 10.0f*zo);
		Cd = 0.4f / (logf(hdrag / zo) - 1.0f);
		Cd = Cd*Cd;

		//Bottom sediment
		//Asb=0.005f*hhi[tx][ty]*powf(D50/hhi[tx][ty]/(drho*g*D50),1.2f);
		Asb = 0.005f*hhi[tx][ty] * powf(1 / hhi[tx][ty] / (drho*g), 1.2f);//simplified from above to limit the propagation of round of error with D50

		//Suspended Sediment
		Ass = 0.012f*D50*powf(dester, -0.6f) / (powf(drho*g*D50, 1.2f));



		//
		T1 = vmags + 0.018f / Cd*Urms2;
		T1 = min(T1, 100000.0f*g / zom[i] * D50*drho);
		T1 = sqrtf(T1);




		// Calculate Cequilibrium


		if (hhi[tx][ty] > eps)
		{
			wetz = 1;
		}
		else
		{
			wetz = 0;
		}


		DECNUM T2;
		T2 = 0.0f;

		if (T1 > Ucr && hhi[tx][ty] > eps)
		{
			T2 = powf(T1 - Ucr, 2.4f);
		}





		ceqb = Asb*T2;
		ceqb = min(ceqb / hhi[tx][ty], 0.05f);             //maximum equilibrium bed concentration

		// This should be different for each sediment fraction
		ceqbg[i] = (1 - bulk)*ceqb*sedcal*wetz;

		ceqs = min(Ass*T2 / hhi[tx][ty], 0.05f);// maximum equilibrium suspended concentration		      

		ceqsg[i] = (ceqs + bulk*ceqb)*sedcal*wetz;
	}
}


__global__ void Sednew(int nx, int ny, DECNUM rho,DECNUM g,DECNUM eps, DECNUM Trep,DECNUM D50, DECNUM D90, DECNUM rhosed,DECNUM ws,DECNUM nuhfac,DECNUM * ueu, DECNUM * vev,DECNUM *H,DECNUM * DR,DECNUM * R, DECNUM * c,DECNUM * hh,DECNUM *urms,DECNUM * ceqsg,DECNUM * ceqbg, DECNUM *Tsg, DECNUM *zom, DECNUM * kturb)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i=ix+iy*nx;
	int tx =threadIdx.x;
	int ty= threadIdx.y;




	

	__shared__ DECNUM  hhi[16][16];
	//__shared__ DECNUM  Hi[16][16];
	__shared__ DECNUM  ueui[16][16];
	__shared__ DECNUM  ueul[16][16];
	__shared__ DECNUM  vevi[16][16];
	__shared__ DECNUM  vevb[16][16];
	
	DECNUM ue,ve;
	
	DECNUM vmags,vmag,ML,Tbore,dcfin,dcf,kb,Urms2;
	DECNUM B2,T1,Ucrc,Ucrw,Ucr,Ass,Asb,Cd,ceqb,ceqs;
	//DECNUM D50=0.0038;
	//DECNUM D90=0.0053;

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);
		DECNUM zo = zom[i];
		DECNUM sedcal = 1.0f;
		int wetz;
		DECNUM bulk = 1.0f;

		//DECNUM rhosed=25000; //Sediment density
		DECNUM drho = (rhosed - rho) / rho;
		DECNUM dester = powf(drho*g, 1.0f / 3.0f) / 0.0001*D50;
		// Calc euler velocities at cell center
		hhi[tx][ty] = hh[i];
		ueui[tx][ty] = ueu[i];
		vevi[tx][ty] = vev[i];
		//Hi[tx][ty]=H[i];



		ueul[tx][ty] = ueu[xminus + iy*nx];
		vevb[tx][ty] = vev[ix + yminus*nx];
		__syncthreads;

		ue = 0.5*(ueul[tx][ty] + ueui[tx][ty]);
		ve = 0.5*(vevb[tx][ty] + vevi[tx][ty]);

		//need to check this...
		vmags = ue*ue + ve*ve;
		vmag = sqrt(vmags);

		//Mixing length
		ML = max(min(sqrt(2 * R[i] * Trep / (rho*c[i])), hhi[tx][ty]), 0.01f);

		//Bore period
		Tbore = Trep / 4.0f;// should be more complex //to improve later

		//Exponential decay of turbulence over time
		dcfin = exp(min(hhi[tx][ty], 100.0f) / ML);
		dcf = min(1 / (dcfin - 1), 1.0f);

		//Short wave turbulence (Breaking):
		kb = nuhfac*powf(DR[i] / rho, 0.6666667f)*dcf;

		Urms2 = urms[i] * urms[i] + 1.45f*(kb + kturb[i]);//not been tested yet!!!
		//DECNUM dester=rhosed*D50;//dester=25296*D50;
		//DECNUM dster=(drho*g/1.0f-12)**onethird*s%D50(jg) 
		DECNUM tsfac = 0.1f;
		//DECNUM ws=0.043f;
		DECNUM Tsmin = 0.5f;
		Tsg[i] = max(tsfac*hhi[tx][ty] / ws, Tsmin); //should be different for each sediment class

		//DECNUM Ucrc,Ucrw;
		if (D50 <= 0.0005)
		{
			Ucrc = powf(0.19f*D50, 0.10f)*log10f(4.0f*hhi[tx][ty] / D90);
			Ucrw = powf(0.24f*drho*g, 0.66f)*powf(D50*Trep, 0.33);
		}
		if (D50<0.002 && D50>0.0005)
		{
			//critical U due to current
			Ucrc = 8.5f*powf(D50, 0.6f)*log(4.0f*hhi[tx][ty] / D90) / log(10.0f);//Shields

			//Critical U due to Waves
			Ucrw = 0.95f*powf(1.65f*g, 0.57f)*powf(D50, 0.43f)*powf(Trep, 0.14f);//Komar and Miller 1975
		}


		B2 = vmag / max(vmag + sqrtf(Urms2), eps);
		//Critical velocity
		Ucr = B2*Ucrc + (1.0f - B2)*Ucrw;

		//Bottom sediment
		Asb = 0.015f*hhi[tx][ty] * powf(D50 / hhi[tx][ty], 1.2f) / powf(drho*g*D50, 0.75f);

		//Suspended Sediment
		Ass = 0.012f*D50*powf(dester, -0.6f) / (powf(drho*g*D50, 1.2f));

		//
		T1 = vmags + 0.64f*Urms2;

		T1 = min(T1, 100000 * g / zom[i] * D50*drho);
		T1 = sqrtf(T1);


		// Calculate Cequilibrium


		if (hhi[tx][ty] > eps)
		{
			wetz = 1;
		}
		else
		{
			wetz = 0;
		}


		DECNUM T2;
		T2 = 0.0f;

		if (T1 > Ucr && hhi[tx][ty] > eps)
		{
			T2 = powf((T1 - Ucr), 1.5f);

			ceqb = Asb*T2;
			ceqb = min(ceqb / hhi[tx][ty], 0.05f);             //maximum equilibrium bed concentration
			// This should be different for each sediment fraction
			T2 = powf((T1 - Ucr), 2.4f);
			ceqs = min(Ass*T2 / hhi[tx][ty], 0.05f);// maximum equilibrium suspended concentration		      
		}

		ceqbg[i] = (1 - bulk)*ceqb*sedcal*wetz;
		ceqsg[i] = (ceqs + bulk*ceqb)*sedcal*wetz;
	}
}



__global__ void Rvr(int nx, int ny,DECNUM Trep,DECNUM facsk,DECNUM facas,DECNUM * H, DECNUM * hh, DECNUM * urms, DECNUM * c, DECNUM *ua)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i=ix+iy*nx;


	if (ix < nx && iy < ny)
	{
		// time averaged flows due to wave asymmetry

		DECNUM m1 = 0.0f;       // a = 0
		DECNUM m2 = 0.7939f;  // b = 0.79 +/- 0.023
		DECNUM m3 = -0.6065f; // c = -0.61 +/- 0.041
		DECNUM m4 = 0.3539f;  // d = -0.35 +/- 0.032 
		DECNUM m5 = 0.6373f;  // e = 0.64 +/- 0.025
		DECNUM m6 = 0.5995f;  // f = 0.60 +/- 0.043
		DECNUM alpha = -1.0f*log10(exp(1.0f)) / m4;
		DECNUM beta = exp(m3 / m4);
		DECNUM k = 2 * pi / (c[i] * Trep);

		DECNUM Ur, Bm, B1, Sk, As;
		//	if (abs(facua)>0.d0f)
		//	{
		Ur = 3.0f / 8.0f*sqrt(2.0f)*H[i] * k / powf(k*hh[i], 3.0f);                  //Ursell number
		Ur = max(Ur, 0.00001f);
		Bm = m1 + (m2 - m1) / (1.0f + beta*powf(Ur, alpha));                    //Boltzmann sigmoid (eq 6)         
		B1 = (-90.0f + 90.0f*tanh(m5 / powf(Ur, m6)))*pi / 180.0f;
		Sk = Bm*cos(B1);                                            //Skewness (eq 8)
		As = Bm*sin(B1);                                            //Asymmetry(eq 9)
		ua[i] = (facsk*Sk - facas*As)*urms[i];
		//	}
	}
}

__global__ void Erosus(int nx, int ny, DECNUM dt,DECNUM morfac,DECNUM por ,DECNUM * hh,DECNUM * ceqsg,DECNUM * ceqbg, DECNUM *Tsg, DECNUM * facero, DECNUM * structdepth)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i=ix+iy*nx;
	

	if (ix < nx && iy < ny)
	{
		//DECNUM morfac=0.0f; //Morphological factor 0.0= no bed update 1.0= normal bed update >1.0= enhance bed update
		//DECNUM por=0.4f;
		DECNUM pbbed = 1.0f;// sand fraction everywhere
		DECNUM exp_ero;
		//to be done for each sediment class


		exp_ero = morfac*dt / (1.0f - por)*hh[i] * (ceqsg[i] * pbbed / Tsg[i] + ceqbg[i] * pbbed / dt);
		facero[i] = min(1.0f, structdepth[i] * pbbed / max(0.000001f, exp_ero));        // limit erosion to available sediment on top 
	}
}


__global__ void Susp(int nx, int ny,DECNUM dx, DECNUM eps, DECNUM nuh,DECNUM nuhfac, DECNUM rho,DECNUM sus,DECNUM bed,DECNUM * ueu,DECNUM * vev,DECNUM * uu,DECNUM * uvg,DECNUM * hug,DECNUM * vv,DECNUM *vug,DECNUM *hvg,DECNUM * zb,DECNUM *h,DECNUM * DR, DECNUM * C,DECNUM * ceqbg,DECNUM * Sus, DECNUM * Svs,DECNUM * Sub, DECNUM * Svb,DECNUM * thetamean,DECNUM * ua)
{

	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i=ix+iy*nx;
	int tx =threadIdx.x;
	int ty= threadIdx.y;


	DECNUM cu,cv,Dc,dcsdx,dcsdy,hu,hv;
	DECNUM dzbdx,dzbdy;
	DECNUM wetu,wetv;
	
	DECNUM vmagu, vmagv;
	DECNUM uau,uav;
	DECNUM uv,vu;
	DECNUM cub,cvb;
	
	//DECNUM sus=1.0f;
	//DECNUM bed=1.0f;
	
	DECNUM pbbed=1.0f; // WARNING sand fraction every where
	
	DECNUM facsl=1.6f; // between 0 and 1.6 tke into account the bed slope in bed load calculations
	DECNUM urep,vrep;



	
	__shared__ DECNUM  cci[16][16];
	__shared__ DECNUM  ccr[16][16];
	__shared__ DECNUM  cct[16][16];


	__shared__ DECNUM  cbi[16][16];
	__shared__ DECNUM  cbr[16][16];
	__shared__ DECNUM  cbt[16][16];

	__shared__ DECNUM hhi[16][16];
	
	__shared__ DECNUM zbi[16][16];
	__shared__ DECNUM zbr[16][16];
	__shared__ DECNUM zbt[16][16];
	
	__shared__ DECNUM uui[16][16];

	
	__shared__ DECNUM vvi[16][16];


	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);



		hhi[tx][ty] = h[i];


		zbi[tx][ty] = zb[i];
		zbr[tx][ty] = zb[xplus + iy*nx];
		zbt[tx][ty] = zb[ix + yplus*nx];


		cci[tx][ty] = C[i];
		ccr[tx][ty] = C[xplus + iy*nx];

		cct[tx][ty] = C[ix + yplus*nx];

		cbi[tx][ty] = pbbed*ceqbg[i];
		cbr[tx][ty] = pbbed*ceqbg[xplus + iy*nx];

		cbt[tx][ty] = pbbed*ceqbg[ix + yplus*nx];

		uui[tx][ty] = uu[i];


		vvi[tx][ty] = vv[i];



		__syncthreads;

		uau = 0.5*cosf(thetamean[i])*(ua[i] + ua[xplus + iy*nx]);
		uav = 0.5*sinf(thetamean[i])*(ua[xplus + iy*nx] + ua[i]);

		uv = uvg[i];//0.25f*(uul[tx][ty]+uui[tx][ty]+uutl[tx][ty]+uut[tx][ty]);
		vu = vug[i];//0.25f*(vvb[tx][ty]+vvi[tx][ty]+vvbr[tx][ty]+vvr[tx][ty]);
		urep = ueu[i] + uau;
		vmagu = sqrtf(powf((uui[tx][ty] + uau), 2.0f) + powf((vu + uav), 2.0f));

		uau = 0.5*cosf(thetamean[i])*(ua[i] + ua[ix + yplus*nx]);
		uav = 0.5*sinf(thetamean[i])*(ua[ix + yplus*nx] + ua[i]);

		vrep = vev[i] + uav;
		vmagv = sqrtf(powf(uv + uau, 2.0f) + powf(vvi[tx][ty] + uav, 2.0f));

		dzbdx = -1.0f*(zbr[tx][ty] - zbi[tx][ty]) / dx;
		dzbdy = -1.0f*(zbt[tx][ty] - zbi[tx][ty]) / dx;


		hu = hug[i];//0.50f*(hhi[tx][ty]+hhr[tx][ty]);
		hv = hvg[i];//0.50f*(hhi[tx][ty]+hht[tx][ty]);

		wetu = 0.0f;
		wetv = 0.0f;

		if (hu > eps)
		{
			wetu = 1.0f;
		}
		if (hv > eps)
		{
			wetv = 1.0f;
		}






		if (urep > 0.0f)
		{

			cu = cci[tx][ty];
			cub = cbi[tx][ty];
		}
		else
		{
			if (urep < 0.0f)
			{
				cu = ccr[tx][ty];
				cub = cbr[tx][ty];
			}
			else
			{
				cu = 0.50f*(cci[tx][ty] + ccr[tx][ty]);
				cub = 0.50f*(cbi[tx][ty] + cbr[tx][ty]);
			}
		}
		dcsdx = (ccr[tx][ty] - cci[tx][ty]) / dx;

		if (vrep > 0.0f)
		{

			cv = cci[tx][ty];
			cvb = cbi[tx][ty];
			//cvb(i,j)=par%thetanum*pbbed(i,j,1,jg)*ceqbg(i,j,jg)+(1.d0-par%thetanum)*pbbed(i,min(j+1,ny),1,jg)*ceqbg(i,min(j+1,ny),jg)
		}
		else
		{
			if (vrep < 0.0f)
			{
				cv = cct[tx][ty];
				cvb = cbt[tx][ty];
				//cvb(i,j)=par%thetanum*pbbed(i,j+1,1,jg)*ceqbg(i,j+1,jg)+(1.d0-par%thetanum)*pbbed(i,max(j,2),1,jg)*ceqbg(i,max(j,2),jg)
			}
			else
			{
				cv = 0.50f*(cci[tx][ty] + cct[tx][ty]);
				cvb = 0.50f*(cbi[tx][ty] + cbt[tx][ty]);
				//cvb(i,j)=0.5d0*(pbbed(i,j,1,jg)*ceqbg(i,j,jg)+pbbed(i,j+1,1,jg)*ceqbg(i,j+1,jg))

			}
		}

		dcsdy = (cct[tx][ty] - cci[tx][ty]) / dx;

		Dc = nuh + nuhfac*hhi[tx][ty] * powf(DR[i] / rho, 1.0f / 3.0f);

		Sus[i] = sus*(cu*urep*hu - Dc*hu*dcsdx - facsl*cu*vmagu*hu*dzbdx)*wetu;

		Svs[i] = sus*(cv*vrep*hv - Dc*hv*dcsdy - facsl*cv*vmagv*hv*dzbdy)*wetv;

		Sub[i] = bed*(cub*urep*hu - facsl*cub*vmagu*hu*dzbdx)*wetu;

		Svb[i] = bed*(cvb*vrep*hv - facsl*cvb*vmagv*hv*dzbdy)*wetv;
	}

}



__global__ void Conc(int nx, int ny, DECNUM dx, DECNUM dt,DECNUM eps,DECNUM * hh,DECNUM * C, DECNUM * ceqsg, DECNUM *Tsg,DECNUM *facero,DECNUM * ero,DECNUM * depo,DECNUM * Sus,DECNUM *Svs)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i=ix+iy*nx;
	int tx =threadIdx.x;
	int ty= threadIdx.y;



	__shared__ DECNUM  Susi[16][16];
	__shared__ DECNUM  Susl[16][16];
	__shared__ DECNUM  Svsi[16][16];
	__shared__ DECNUM  Svsb[16][16];
	__shared__ DECNUM  hhi[16][16];

	DECNUM cs,dsusdx,dsvsdy,wetz;
	DECNUM Pbed=1.0f;

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);
		hhi[tx][ty] = hh[i];

		Susi[tx][ty] = Sus[i];
		Susl[tx][ty] = Sus[xminus + iy*nx];
		Svsi[tx][ty] = Svs[i];
		Svsb[tx][ty] = Svs[ix + yminus*nx];

		__syncthreads;

		wetz = 0.0f;
		if (hhi[tx][ty] > eps)
		{
			wetz = 1.0f;
		}


		ero[i] = facero[i] * hhi[tx][ty] * ceqsg[i] * Pbed / Tsg[i];

		dsusdx = (Susi[tx][ty] - Susl[tx][ty]) / dx;

		dsvsdy = (Svsi[tx][ty] - Svsb[tx][ty]) / dx;

		cs = (dt*Tsg[i]) / (dt + Tsg[i])*(hhi[tx][ty] * C[i] / dt - (dsusdx + dsvsdy - ero[i]))*wetz;
		cs = max(cs, 0.0f);
		cs = min(cs, 0.1f*hhi[tx][ty]);

		depo[i] = cs / Tsg[i];
		//cs=cs/hh[i];



		C[i] = cs / hh[i];
	}
}


__global__ void CClatbnd(int nx, int ny, DECNUM eps, DECNUM * hh, DECNUM * C)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	int tx = threadIdx.x;
	int ty = threadIdx.y;


	__shared__ DECNUM cci[16][16];
	__shared__ DECNUM cct[16][16];
	__shared__ DECNUM ccb[16][16];
	__shared__ DECNUM ccr[16][16];

	//cci[tx][ty]=C[i];
	//cct[tx][ty]=C[ix+yplus*nx];
	//ccb[tx][ty]=C[ix+yminus*nx];
	//ccr[tx][ty]=C[xplus+iy*nx];
	//__syncthreads;
	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		if (iy == 0)
		{
			C[i] = C[ix + yplus*nx];
		}

		if (iy == ny - 1)
		{
			C[i] = C[ix + yminus*nx];
		}
		if (ix == 0)
		{
			C[i] = 0.0f;//ccr[tx][ty];
		}
	}
}

__global__ void hardlayer(int nx, int ny,DECNUM dx,DECNUM dt,DECNUM * Sub, DECNUM * Svb, DECNUM * Sout, int * indSub,int * indSvb)
{

	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i=ix+iy*nx;
	int tx =threadIdx.x;
	int ty= threadIdx.y;


	
	__shared__ DECNUM  Subi[16][16];
	__shared__ DECNUM  Subl[16][16];
	__shared__ DECNUM  Svbi[16][16];
	__shared__ DECNUM  Svbb[16][16];
	__shared__ DECNUM  Souti[16][16];
	
	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);
		Subi[tx][ty] = Sub[i];
		Subl[tx][ty] = Sub[xminus + iy*nx];
		Svbi[tx][ty] = Svb[i];
		Svbb[tx][ty] = Svb[ix + yminus*nx];
		Souti[tx][ty] = 0.0f;
		indSub[i] = 0;
		indSvb[i] = 0;

		if (Subi[tx][ty] > 0.0f) //      ! bed load u-direction
		{
			indSub[i] = 1;
			Souti[tx][ty] = Souti[tx][ty] + Subi[tx][ty] * dx;
		}
		if (Svbi[tx][ty] > 0.0f) //     ! bed load v-direction
		{
			indSvb[i] = 1;
			Souti[tx][ty] = Souti[tx][ty] + Svbi[tx][ty] * dx;
		}
		// fluxes at i-1,j
		if (Subl[tx][ty] < 0.0f) //   ! bed load u-direction
		{
			Souti[tx][ty] = Souti[tx][ty] - Subl[tx][ty] * dx;
		}
		// fluxes at i,j-1
		if (Svbb[tx][ty] < 0.0f) //   ! bed load v-direction
		{
			Souti[tx][ty] = Souti[tx][ty] - Svbb[tx][ty] * dx;
		}


		Sout[i] = Souti[tx][ty];
	}
}

__global__ void bedupdate(int nx, int ny, DECNUM eps, DECNUM dx, DECNUM dt, DECNUM morfac, DECNUM por, DECNUM * hh, DECNUM * ero, DECNUM * depo, DECNUM * Sub, DECNUM * Svb, DECNUM * Sout, int * indSub, int * indSvb, DECNUM * zb, DECNUM *ddzb, DECNUM * structdepth)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = ix + iy*nx;
	int tx = threadIdx.x;
	int ty = threadIdx.y;





	__shared__ DECNUM  Subi[16][16];
	__shared__ DECNUM  Subl[16][16];
	__shared__ DECNUM  Svbi[16][16];
	__shared__ DECNUM  Svbb[16][16];

	__shared__ int indSubi[16][16];
	__shared__ int indSubl[16][16];
	__shared__ int indSvbi[16][16];
	__shared__ int indSvbb[16][16];

	if (ix < nx && iy < ny)
	{

		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		DECNUM oldzb = zb[i];
		DECNUM fac;
		DECNUM Savailable;
		DECNUM pbbed = 1.0f;

		Subi[tx][ty] = Sub[i];
		Subl[tx][ty] = Sub[xminus + iy*nx];
		Svbi[tx][ty] = Svb[i];
		Svbb[tx][ty] = Svb[ix + yminus*nx];

		indSubi[tx][ty] = indSub[i];
		indSubl[tx][ty] = indSub[xminus + iy*nx];
		indSvbi[tx][ty] = indSvb[i];
		indSvbb[tx][ty] = indSvb[ix + yminus*nx];


		__syncthreads;


		Savailable = structdepth[i] * pbbed / morfac / dt*(1.0f - por)*dx*dx;

		//	 ! reduction factor for cell outgoing sediment transports

		fac = 1.0f;
		if (Sout[i] > 0.0f)
		{
			fac = min(1.0f, Savailable / Sout[i]);
		}

		if (fac < 1.0f)
		{
			Subi[tx][ty] = fac*indSubi[tx][ty] * Subi[tx][ty] + (1 - indSubi[tx][ty])*Subi[tx][ty];
			Subl[tx][ty] = fac*(1 - indSubl[tx][ty])*Subl[tx][ty] + indSubl[tx][ty] * Subl[tx][ty];
			Svbi[tx][ty] = fac*indSvbi[tx][ty] * Svbi[tx][ty] + (1 - indSvbi[tx][ty])*Svbi[tx][ty];
			Svbb[tx][ty] = fac*(1 - indSvbb[tx][ty])*Svbb[tx][ty] + indSvbb[tx][ty] * Svbb[tx][ty];
		}




		DECNUM dzg;


		dzg = morfac*dt / (1.0f - por)*(ero[i] - depo[i] /*+ (Subi[tx][ty]-Subl[tx][ty])/dx + (Svbi[tx][ty]-Svbb[tx][ty])/dx*/);


		zb[i] = zb[i] + dzg;
		hh[i] = hh[i] + dzg;
		ddzb[i] = -1 * dzg;
		structdepth[i] = structdepth[i] - dzg;


	}
}





__global__ void zblatbnd(int nx,int ny,DECNUM * F)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i=ix+iy*nx;
	int tx =threadIdx.x;
	int ty= threadIdx.y;

	
	
	
	//__shared__ DECNUM Fi[16][16];
	//__shared__ DECNUM Ft[16][16];
	//__shared__ DECNUM Fb[16][16];
	//__shared__ DECNUM Fr[16][16];
	
	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);

		//Fi[tx][ty]=F[i];
		//Ft[tx][ty]=
		//Fb[tx][ty]=F[ix+yminus*nx];
		//Fr[tx][ty]=F[xplus+iy*nx];
		__syncthreads;

		//F[i]=Fi[tx][ty];
		if (iy == 0)
		{
			F[i] = F[ix + yplus*nx];
		}
		if (iy == ny - 1)
		{
			F[i] = F[ix + yminus*nx];
		}
		if (ix == 0)
		{
			F[i] = F[xplus + iy*nx];
		}

	}
			



}

__global__ void avalanching(int nx, int ny,DECNUM eps,DECNUM dx,DECNUM dt,DECNUM por,DECNUM drydzmax,DECNUM wetdzmax,DECNUM maxslpchg,DECNUM * hh,DECNUM * zb,DECNUM * dzb,DECNUM * structdepth)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i=ix+iy*nx;
	int tx =threadIdx.x;
	int ty= threadIdx.y;

	
	__shared__ DECNUM Zbi[16][16];
	__shared__ DECNUM Zbt[16][16];
	__shared__ DECNUM Zbr[16][16];
	
	__shared__ DECNUM dzbi[16][16];
	__shared__ DECNUM dzbt[16][16];
	__shared__ DECNUM dzbr[16][16];
	
	__shared__ DECNUM stdepi[16][16];
	__shared__ DECNUM stdept[16][16];
	__shared__ DECNUM stdepr[16][16];
	

	if (ix < nx && iy < ny)
	{
		unsigned int xminus = mminus(ix, nx);
		unsigned int xplus = pplus(ix, nx);
		unsigned int yminus = mminus(iy, ny);
		unsigned int yplus = pplus(iy, ny);


		Zbi[tx][ty] = zb[i];
		Zbt[tx][ty] = zb[ix + yplus*nx];
		Zbr[tx][ty] = zb[xplus + iy*nx];

		dzbi[tx][ty] = dzb[i];
		dzbt[tx][ty] = dzb[ix + yplus*nx];
		dzbr[tx][ty] = dzb[xplus + iy*nx];

		stdepi[tx][ty] = structdepth[i];
		stdept[tx][ty] = structdepth[ix + yplus*nx];
		stdepr[tx][ty] = structdepth[xplus + iy*nx];
		__syncthreads;

		DECNUM dzmaxdry = drydzmax;//1.0; // critical avalanching slope above water (dzbdx)
		DECNUM dzmaxwet = wetdzmax;//0.3; // critical avalanching slope under water
		DECNUM maxchg = maxslpchg;//0.05; // 0.05max bedlavel change due to Avalanching in m/s/m This avoid generatng tsunamis from avalanching
		DECNUM dzbdx, dzbdy, dzmax, dzbdxsign, dzbdysign;
		DECNUM dzbx = 0.0f;
		DECNUM dzby = 0.0f;

		if (hh[i] > eps)
		{
			dzmax = dzmaxwet;
		}
		else
		{
			dzmax = dzmaxdry;
		}


		// X direction
		dzbdx = (Zbr[tx][ty] - Zbi[tx][ty]) / dx;
		if (fabs(dzbdx) > dzmax)
		{
			dzbdxsign = dzbdx / fabs(dzbdx);
			dzbx = dzbdxsign*(fabs(dzbdx) - dzmax)*dx;
			if (dzbdxsign > 0)
			{
				dzbx = min(dzbx, maxchg*dt / dx);
				dzbx = min(dzbx, stdepi[tx][ty]);
			}
			else
			{
				dzbx = max(dzbx, -1.0f*maxchg*dt / dx);
				dzbx = max(dzbx, -1.0f*stdepr[tx][ty]);

			}
		}






		// Y direction
		dzbdy = (Zbt[tx][ty] - Zbi[tx][ty]) / dx;
		if (abs(dzbdy) > dzmax)
		{
			dzbdysign = dzbdy / abs(dzbdy);
			dzby = dzbdysign*(abs(dzbdy) - dzmax)*dx;
			if (dzbdysign > 0)
			{
				dzby = min(dzby, maxchg*dt / dx);
				dzby = min(dzby, stdepi[tx][ty]);
			}
			else
			{
				dzby = max(dzby, -1 * maxchg*dt / dx);
				dzby = max(dzby, -1 * stdept[tx][ty]);

			}
		}

		dzbi[tx][ty] = dzbi[tx][ty] + dzbx + dzby;
		dzbr[tx][ty] = dzbr[tx][ty] - dzbx;
		dzbt[tx][ty] = dzbt[tx][ty] - dzby;
		//__syncthreads;

		//Zb[i]=zbi[tx][ty]+dzbi[tx][ty]+dzbr[tx-1][ty]+dzbt[tx][ty-1];
		dzb[i] = dzb[i] + dzbx + dzby;
		dzb[xplus + iy*nx] = dzb[xplus + iy*nx] - dzbx;

		dzb[ix + yplus*nx] = dzb[ix + yplus*nx] - dzby;

	}


}

__global__ void updatezb(int nx,int ny,DECNUM dx,DECNUM dt,DECNUM * zb,DECNUM * ddzb,DECNUM * dzb,DECNUM * zs,DECNUM *hh, DECNUM * structdepth)
						
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i=ix+iy*nx;
	
	int tx =threadIdx.x;
	int ty= threadIdx.y;


	if (ix < nx && iy < ny)
	{

		zb[i] = zb[i] + ddzb[i];
		dzb[i] = dzb[i] - ddzb[i];
		zs[i] = zs[i] - ddzb[i];
		structdepth[i] = structdepth[i] - ddzb[i];
		hh[i] = hh[i] - ddzb[i];
	}
}


__global__ void updatezom(int nx, int ny,DECNUM cf,DECNUM cf2,DECNUM fw,DECNUM fw2,DECNUM * structdepth, DECNUM * cfm,DECNUM * fwm)
{
	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i=ix+iy*nx;
	
	int tx =threadIdx.x;
	int ty= threadIdx.y;
	
	
	if (ix < nx && iy < ny)
	{

		if (structdepth[i] < 0.05f)
		{
			cfm[i] = cf2;
			fwm[i] = fw2;
		}
		else
		{
			cfm[i] = cf;
			fwm[i] = fw;
		}
	}
	
	
}



//__global__ void SedEnt(int nx, int ny,DECNUM dx,DECNUM dt, DECNUM rho,DECNUM g,DECNUM eps, DECNUM Trep,DECNUM * ueu, DECNUM * vev,DECNUM *H, DECNUM *DR,DECNUM *R,DECNUM *c,DECNUM * hh,DECNUM *urms,DECNUM *Sedup)
//{
//	unsigned int ix = blockIdx.x*blockDim.x + threadIdx.x;
//	unsigned int iy = blockIdx.y*blockDim.y + threadIdx.y;
//	unsigned int i=ix+iy*nx;
//	int tx =threadIdx.x;
//	int ty= threadIdx.y;
//
//
//	DECNUM Urms2,kb,teta,tau,Ab,fw,abkb,fent;
//
//	DECNUM D50=0.0038;
//	DECNUM D90=0.0090;
//	DECNUM zo=0.05;
//	DECNUM ggm=0.05;
//
//    DECNUM ws=0.043f;
//	DECNUM wp=0.00001f;
//	
//	DECNUM rhosed=25000; //Sediment density
//	DECNUM drho = (rhosed-rho)/rho;
//
//
//	//Short wave turbulence (Breaking):
//	kb=pow(DR[i]/rho,0.6666667f)*1/(exp(min(hh[i]/max(H[i],0.1f),100.0f))-1);
//	
//	Urms2=urms[i]*urms[i]+0.5f*kb;
//	Ab=urms[i]*Trep/(2*pi);
//	kb=30*zo;
//	abkb=Ab/kb;
//
//	if(abkb<=0.2f)
//	{
//		fw=0.3f;
//	}
//	else
//	{
//		if(abkb<=100)
//		{
//			fw=exp(-8.82+7.02*pow(abkb,-0.078f));
//		}
//		else
//		{
//			fw=exp(-7.30+5.61*pow(abkb,-0.109f));
//		}
//	}
//
//
//	tau=max(0.5*fw*rho*Urms2,(ueu[i]*ueu[i]+vev[i]*vev[i])*0.41f*0.41f/pow(log(hh[i]/(3*zo)),2));
//
//	teta=tau/((rhosed-rho)*g*D50);
//
//	fent=ggm*pow(teta,3)*ws;
//	
//	Sedup[i]=round((fent*dt*dx*dx)/wp);
//
//
//
//
//
//}
//
//__global__ void up3ddGPU(int nx, DECNUM *uu,DECNUM *vv,DECNUM *xx, DECNUM *yy, DECNUM *zz, DECNUM *dd_rand, DECNUM dx, DECNUM dt)
//{
//      DECNUM Dpx=10.0;
//      DECNUM Ux=0.05;
//      DECNUM Vx=0.05;
//     
//      //__shared__ DECNUM xxx[256];
//      //__shared__ DECNUM yyy[256];
//      DECNUM xxx,yyy;
//	  
// 
//      
// 
// 
//      
//      DECNUM Eh=0.001;//m2/s
//      DECNUM Ev=0.001;//m2/s
//      DECNUM ws=-0.043f;
//	  //DECNUM ws=-0.1;//m/s
//      int a=0;//abitrary number
//      //DECNUM dt=1;
//      DECNUM zo=0.001;//m roughness length
//      DECNUM ttc=0.01;// critical resuspension velocity m/s
//     
//      int i = blockIdx.x * blockDim.x * blockDim.y + blockDim.x * threadIdx.y + threadIdx.x;
//      int tx = threadIdx.x;
//      int idx = threadIdx.x + blockIdx.x*blockDim.x;
// 
//      xxx=xx[i];
//      yyy=yy[i];
//
//      //DECNUM xp=xxx/dx;
//      //DECNUM yp=yyy/dx;
//      
//      //int x1=floor(xxx/dx);
//      //int y1=floor(yyy/dx);
//      
//      //int x2=x1+1;
//      //int y2=y1+1;
//
//      //DECNUM den=(x2-x1)*(y2-y1);
//      //DECNUM U11,U12,U21,U22,V11,V12,V21,V22;
//      
//      //U11=uu[x1+y1*nx];
//      //U21=uu[x2+y1*nx];
//      //U12=uu[x1+y2*nx];
//      //U22=uu[x2+y2*nx];
//
//      //V11=vv[x1+y1*nx];
//      //V21=vv[x2+y1*nx];
//      //V12=vv[x1+y2*nx];
//      //V22=vv[x2+y2*nx];
//
//
//
// 
//           
//      //Interpolate wter depth, Uvel Vvel at the particle position
//     
//      //Dpx=tex2D(texZ, xxx[tx]/dx, yyy[tx]/dx);
//     
//      Ux=tex2D(texU, xxx/dx, yyy/dx);
//      Vx=tex2D(texV, xxx/dx, yyy/dx);
//	      
//	//Ux=U11;//den*(x2-xp)*(y2-yp)+U21/den*(xp-x1)*(y2-yp)+U12/den*(x2-xp)*(yp-y1)+U22/den*(xp-x1)*(yp-y1);
//	//Vx=V11;//den*(x2-xp)*(y2-yp)+V21/den*(xp-x1)*(y2-yp)+V12/den*(x2-xp)*(yp-y1)+V22/den*(xp-x1)*(yp-y1);
//      //DECNUM T=9.81*1021*(Ux*Ux+Vx*Vx)/(18*log10(0.37*Dpx/zo));//Warning doggy equation!!!!!
//      
//     
//     
// 
//      //update the particle position
//     
//      //if (zz[i]*Dpx>zo)
//      //{
//         
//           
//      //Ux=Ux*(log10f(Dpx*zz[i]/zo)/log10f(0.37*Dpx/zo));
//      //Vx=Vx*(log10f(Dpx*zz[i]/zo)/log10f(0.37*Dpx/zo));
//     
// 
//      xx[i]=xxx+Ux*dt/*+(dd_rand[i]-0.5)*2*sqrtf(6*Eh*dt)*/;
//      yy[i]=yyy+Vx*dt/*+(dd_rand[i+a]-0.5)*2*sqrtf(6*Eh*dt)*/;
//      zz[i] =(zz[i]*Dpx+ws*dt/*+(dd_rand[i+2*a]-0.5)*2*sqrtf(6*Ev*dt)*/)/Dpx;
//
//      
//      
//      
//      
//     
// 
//            
// 
// 
//     
//
//
//}
//
